// Copyright (c) 2020 Michael Koesel and respective contributors
// SPDX-License-Identifier: MIT
// See accompanying LICENSE file for detailed information
#include <iostream>
#include "mapping/laser_to_meas_grid.h"
#include "mapping/opengl/renderer.h"

#include "mapping/kernel/measurement_grid.h"

LaserMeasurementGrid::LaserMeasurementGrid(const Params& params, float grid_length, float resolution)
    : grid_size(static_cast<int>(grid_length / resolution)), params(params)
{
    int grid_cell_count = grid_size * grid_size;
    std::cout << "M1" << std::endl;
    CHECK_ERROR(hipMalloc(&meas_grid, grid_cell_count * sizeof(dogm::MeasurementCell)));
    std::cout << "M2" << std::endl;
    std::cout << "grid_size: " << grid_size << ", fov: " << params.fov << ", grid length: " << grid_length << ", max_range" << params.max_range << std::endl;
    renderer = std::make_unique<Renderer>(grid_size, params.fov, grid_length, params.max_range);
    std::cout << "M3" << std::endl;
}

LaserMeasurementGrid::~LaserMeasurementGrid()
{
    CHECK_ERROR(hipFree(meas_grid));
}

dogm::MeasurementCell* LaserMeasurementGrid::generateGrid(const std::vector<float>& measurements)
{
    const int num_measurements = measurements.size();
    std::cout << "M1" << std::endl;
    float* d_measurements;
    CHECK_ERROR(hipMalloc(&d_measurements, num_measurements * sizeof(float)));
    CHECK_ERROR(
        hipMemcpy(d_measurements, measurements.data(), num_measurements * sizeof(float), hipMemcpyHostToDevice));
    std::cout << "M2" << std::endl;	
    const int polar_width = num_measurements;
    const int polar_height = static_cast<int>(params.max_range / params.resolution);

    dim3 dim_block(32, 32);
    dim3 grid_dim(divUp(polar_width, dim_block.x), divUp(polar_height, dim_block.y));
    dim3 cart_grid_dim(divUp(grid_size, dim_block.x), divUp(grid_size, dim_block.y));

    const float anisotropy_level = 16.0f;
    Texture polar_texture(polar_width, polar_height, anisotropy_level);
    hipSurfaceObject_t polar_surface;
    std::cout << "M3" << std::endl;
    // create polar texture
    polar_texture.beginCudaAccess(&polar_surface);
    createPolarGridTextureKernel<<<grid_dim, dim_block>>>(polar_surface, d_measurements, polar_width, polar_height,
                                                          params.resolution);
    std::cout << "M4" << std::endl;
    CHECK_ERROR(hipGetLastError());
    polar_texture.endCudaAccess(polar_surface);

    // render cartesian image to texture using polar texture
    renderer->renderToTexture(polar_texture);
    std::cout << "M5" << std::endl;
    auto framebuffer = renderer->getFrameBuffer();
    hipSurfaceObject_t cartesian_surface;

    framebuffer->beginCudaAccess(&cartesian_surface);
    // transform RGBA texture to measurement grid
    cartesianGridToMeasurementGridKernel<<<cart_grid_dim, dim_block>>>(meas_grid, cartesian_surface, grid_size);
    std::cout << "M6" << std::endl;
    CHECK_ERROR(hipGetLastError());
    framebuffer->endCudaAccess(cartesian_surface);

    CHECK_ERROR(hipFree(d_measurements));
    CHECK_ERROR(hipDeviceSynchronize());
    std::cout << "M7" << std::endl;
    return meas_grid;
}
