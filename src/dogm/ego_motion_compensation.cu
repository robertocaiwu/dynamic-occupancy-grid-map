#include "hip/hip_runtime.h"
// Copyright (c) 2020 Michael Koesel and respective contributors
// SPDX-License-Identifier: MIT
// See accompanying LICENSE file for detailed information

#include "common.h"
#include "cuda_utils.h"
#include "dogm_types.h"
#include "dogm/ego_motion_compensation.h"

#include <hip/hip_runtime.h>
#include <>

namespace dogm
{

__global__ void moveParticlesKernel(ParticlesSoA particle_array, int x_move, int y_move, int particle_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
    {
        particle_array.state[i][0] -= x_move;
        particle_array.state[i][1] -= y_move;
    }
}

__global__ void moveMapKernel(GridCell* __restrict__ grid_cell_array, const GridCell* __restrict__ old_grid_cell_array,
                              int x_move, int y_move, int grid_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < grid_size && y < grid_size)
    {
        int index = x + grid_size * y;
        int new_y = y + y_move;
        int new_x = x + x_move;
        int new_index = new_x + grid_size * new_y;

        if (new_x > 0 && new_x < grid_size && new_y > 0 && new_y < grid_size)
        {
            grid_cell_array[index] = old_grid_cell_array[new_index];
        }
    }
}

} /* namespace dogm */
